#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cassert>
#include <cstdio>
#include "Compack3D_utils_kernels.cuh"
#include "Compack3D_tri_kernels.cuh"

namespace cmpk {
namespace tri {

/*!
 * Solve the local system using PCR along i-dimension
 * \param [in, out] x                   right-hand side of the system and the solution as the output
 * \param [in]      fact_prev           factorization coefficients of the previous row
 * \param [in]      fact_curr           factorization coefficients of the current row
 * \param [in]      fact_next           factorization coefficients of the next row
 * \param [in]      max_sub_sys_size    maximum sub-system size that can be solved with one memory load
 * \param [in]      Ni                  total number of entries in the i-dimension
 * \param [in]      Nj                  total number of entries in the j-dimension
 * \param [in]      Nk                  total number of entries in the k-dimension
 * \param [in]      arr_stride_i        array access stride in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride in j-dimension, from j to j+1
 * \note array access in k-dimension is contiguous
 */
template<typename RealType, unsigned int NUM_THREADS_I, unsigned int NUM_THREADS_K> __global__
void kernelLocalSolTriPCRDimI(
              RealType* __restrict__         x,
        const RealType* __restrict__ fact_prev,
        const RealType* __restrict__ fact_curr,
        const RealType* __restrict__ fact_next,
        const unsigned int    max_sub_sys_size,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
) {
    #define IDX_K            (blockIdx.x * NUM_THREADS_K + (threadIdx.x % NUM_THREADS_K))
    #define IDX_J             blockIdx.y
    #define TID_K            (threadIdx.x % NUM_THREADS_K)
    #define TID_I            (threadIdx.x / NUM_THREADS_K)
    #define NUM_SYS          Ni
    #define SMEM_IDX(I, K)     ((I) * NUM_THREADS_K + (K))
    #define ARR_IDX(I, J, K) ((I) * arr_stride_i + (J) * arr_stride_j + (K))

    assert(blockIdx.z == 0);
    assert((NUM_THREADS_I * NUM_THREADS_K) == blockDim.x);

    extern __shared__ char smem_general[];
    RealType* smem_realtype = reinterpret_cast<RealType*>(smem_general);
    RealType* smem_buf_0 = &smem_realtype[0];
    RealType* smem_buf_1 = &smem_realtype[NUM_SYS * NUM_THREADS_K];
    RealType* x_read  = smem_buf_0;
    RealType* x_write = smem_buf_1;

    const bool VALID_JK = (IDX_J < Nj) && (IDX_K < Nk);

    // Load data to shared memory from global memory
    unsigned int idx_i = TID_I;
    while (idx_i < Ni) {
        if (VALID_JK) x_read[SMEM_IDX(idx_i, TID_K)] = x[ARR_IDX(idx_i, IDX_J, IDX_K)];
        idx_i += NUM_THREADS_I;
    }
    __syncthreads();

    // Parallel cyclic-reduction on shared memory
    int stride = 1;
    int level  = 0;
    while (stride < NUM_SYS) {
        x_read  = (level & 0b1) ? smem_buf_1 : smem_buf_0;
        x_write = (level & 0b1) ? smem_buf_0 : smem_buf_1;

        idx_i = TID_I;
        while (idx_i < Ni) {
            if (VALID_JK) {
                const int i_sub = idx_i >> level;
                const int n_sub = (NUM_SYS + stride - 1 - (idx_i & (stride-1))) >> level;
                const int row_label = 0b010
                                    + ((i_sub >        0 ) << 2)
                                    + ((i_sub < (n_sub-1))     );
                const unsigned int idx_fact = level * NUM_SYS + locFactIdx<unsigned int>(idx_i, NUM_SYS, stride, max_sub_sys_size);
                const unsigned int SMEM_IDX_CURR = SMEM_IDX(idx_i, TID_K);
                x_write[SMEM_IDX_CURR] = fact_curr[idx_fact] * x_read[SMEM_IDX_CURR];
                if (row_label & 0b100) x_write[SMEM_IDX_CURR] += fact_prev[idx_fact] * x_read[SMEM_IDX(idx_i -  stride, TID_K)];
                if (row_label & 0b001) x_write[SMEM_IDX_CURR] += fact_next[idx_fact] * x_read[SMEM_IDX(idx_i +  stride, TID_K)];
            }
            idx_i += NUM_THREADS_I;
        }
        __syncthreads();
        stride <<= 1;
        level ++;
    }
     
    // Write result to global memeory
    idx_i = TID_I;
    while (idx_i < Ni) {
        if (VALID_JK) x[ARR_IDX(idx_i, IDX_J, IDX_K)] = x_write[SMEM_IDX(idx_i, TID_K)];
        idx_i += NUM_THREADS_I;
    }
     
    #undef IDX_K
    #undef IDX_J
    #undef TID_K
    #undef TID_I
    #undef NUM_SYS
    #undef SMEM_IDX
    #undef ARR_IDX
}



/*!
 * Launch "kernelLocalSolTriPCRDimI<RealType, unsigned int, unsigned int>"
 * \param [in, out] x                   right-hand side of the system and the solution as the output
 * \param [in]      fact_prev           factorization coefficients of the previous row
 * \param [in]      fact_curr           factorization coefficients of the current row
 * \param [in]      fact_next           factorization coefficients of the next row
 * \param [in]      max_sub_sys_size    maximum sub-system size that can be solved with one memory load
 * \param [in]      Ni                  total number of entries in the i-dimension
 * \param [in]      Nj                  total number of entries in the j-dimension
 * \param [in]      Nk                  total number of entries in the k-dimension
 * \param [in]      arr_stride_i        array access stride in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride in j-dimension, from j to j+1
 * \note array access in k-dimension is contiguous
 */
template<typename RealType>
void localSolTriPCRDimI(
              RealType*         x,
        const RealType* fact_prev,
        const RealType* fact_curr,
        const RealType* fact_next,
        const unsigned int  max_sub_sys_size,
        const unsigned int Ni, unsigned const int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
) {
    if (Ni <= 256) { // limited by the size of shared memory (16kB)
        constexpr unsigned int BLOCK_SIZE    = 512;
        constexpr unsigned int DATA_SEG      = 64;
        constexpr unsigned int NUM_THREADS_K = DATA_SEG / sizeof(RealType);
        constexpr unsigned int NUM_THREADS_I = BLOCK_SIZE / NUM_THREADS_K;
        const unsigned int smem_size = 2 * DATA_SEG * Ni;
        dim3 grid_size  = dim3((Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, Nj, 1);
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimI<RealType), NUM_THREADS_I, NUM_THREADS_K>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimI<RealType, NUM_THREADS_I, NUM_THREADS_K><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    } else if (Ni <= 512) { // limited by the size of shared memory (16kB)
        constexpr unsigned int BLOCK_SIZE    = 1024;
        constexpr unsigned int DATA_SEG      = 32;
        constexpr unsigned int NUM_THREADS_K = DATA_SEG / sizeof(RealType);
        constexpr unsigned int NUM_THREADS_I = BLOCK_SIZE / NUM_THREADS_K;
        const unsigned int smem_size = 2 * DATA_SEG * Ni;
        dim3 grid_size  = dim3((Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, Nj, 1);
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimI<RealType), NUM_THREADS_I, NUM_THREADS_K>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimI<RealType, NUM_THREADS_I, NUM_THREADS_K><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    } else {
        constexpr unsigned int BLOCK_SIZE    = 1024;
        constexpr unsigned int DATA_SEG      = 8;
        constexpr unsigned int NUM_THREADS_K = DATA_SEG / sizeof(RealType);
        constexpr unsigned int NUM_THREADS_I = BLOCK_SIZE / NUM_THREADS_K;
        const unsigned int smem_size = 2 * DATA_SEG * Ni;
        dim3 grid_size  = dim3((Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, Nj, 1);
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimI<RealType), NUM_THREADS_I, NUM_THREADS_K>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimI<RealType, NUM_THREADS_I, NUM_THREADS_K><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    }
}



/*!
 * Solve the local system using PCR along j-dimension
 * \param [in, out] x                   right-hand side of the system and the solution as the output
 * \param [in]      fact_prev           factorization coefficients of the previous row
 * \param [in]      fact_curr           factorization coefficients of the current row
 * \param [in]      fact_next           factorization coefficients of the next row
 * \param [in]      max_sub_sys_size    maximum sub-system size that can be solved with one memory load
 * \param [in]      Ni                  total number of entries in the i-dimension
 * \param [in]      Nj                  total number of entries in the j-dimension
 * \param [in]      Nk                  total number of entries in the k-dimension
 * \param [in]      arr_stride_i        array access stride in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride in j-dimension, from j to j+1
 * \note array access in k-dimension is contiguous
 */
template<typename RealType, unsigned int NUM_THREADS_J, unsigned int NUM_THREADS_K> __global__
void kernelLocalSolTriPCRDimJ(
              RealType* __restrict__         x,
        const RealType* __restrict__ fact_prev,
        const RealType* __restrict__ fact_curr,
        const RealType* __restrict__ fact_next,
        const unsigned int      max_sub_sys_size,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
) {
    #define IDX_K            (blockIdx.x * NUM_THREADS_K + (threadIdx.x % NUM_THREADS_K))
    #define IDX_I             blockIdx.z
    #define TID_K            (threadIdx.x % NUM_THREADS_K)
    #define TID_J            (threadIdx.x / NUM_THREADS_K)
    #define NUM_SYS           Nj
    #define SMEM_IDX(J, K)   ((J) * NUM_THREADS_K + (K))
    #define ARR_IDX(I, J, K) ((I) * arr_stride_i + (J) * arr_stride_j + (K))

    assert(blockIdx.y == 0);
    assert((NUM_THREADS_J * NUM_THREADS_K) == blockDim.x);

    extern __shared__ char smem_general[];
    RealType* smem_realtype = reinterpret_cast<RealType*>(smem_general);
    RealType* smem_buf_0 = &smem_realtype[0];
    RealType* smem_buf_1 = &smem_realtype[NUM_SYS * NUM_THREADS_K];
    RealType* x_read  = smem_buf_0;
    RealType* x_write = smem_buf_1;

    const bool VALID_IK = (IDX_K < Nk) && (IDX_I < Ni);

    // Load data to shared memory from global memory
    unsigned int idx_j = TID_J;
    while (idx_j < Nj) {
        if (VALID_IK) x_read[SMEM_IDX(idx_j, TID_K)] = x[ARR_IDX(IDX_I, idx_j, IDX_K)];
        idx_j += NUM_THREADS_J;
    }
    __syncthreads();

    // Parallel cyclic-reduction on shared memory
    int stride = 1;
    int level  = 0;
    while (stride < NUM_SYS) {
        x_read  = (level & 0b1) ? smem_buf_1 : smem_buf_0;
        x_write = (level & 0b1) ? smem_buf_0 : smem_buf_1;

        idx_j = TID_J;
        while (idx_j < Nj) {
            if (VALID_IK) {
                const int j_sub = idx_j >> level;
                const int n_sub = (NUM_SYS + stride - 1 - (idx_j & (stride-1))) >> level;
                const int row_label = 0b010
                                    + ((j_sub >        0 ) << 2)
                                    + ((j_sub < (n_sub-1))     );
                const unsigned int idx_fact = level * NUM_SYS + locFactIdx<unsigned int>(idx_j, NUM_SYS, stride, max_sub_sys_size);
                const unsigned int SMEM_IDX_CURR = SMEM_IDX(idx_j, TID_K);
                x_write[SMEM_IDX_CURR] = fact_curr[idx_fact] * x_read[SMEM_IDX_CURR];
                if (row_label & 0b100) x_write[SMEM_IDX_CURR] += fact_prev[idx_fact] * x_read[SMEM_IDX(idx_j -  stride, TID_K)];
                if (row_label & 0b001) x_write[SMEM_IDX_CURR] += fact_next[idx_fact] * x_read[SMEM_IDX(idx_j +  stride, TID_K)];
            }
            idx_j += NUM_THREADS_J;
        }
        __syncthreads();
        stride <<= 1;
        level ++;
    }
     
    // Write result to global memeory
    idx_j = TID_J;
    while (idx_j < Nj) {
        if (VALID_IK) x[ARR_IDX(IDX_I, idx_j, IDX_K)] = x_write[SMEM_IDX(idx_j, TID_K)];
        idx_j += NUM_THREADS_J;
    }
     
    #undef IDX_K
    #undef IDX_I
    #undef TID_K
    #undef TID_J
    #undef NUM_SYS
    #undef SMEM_IDX
    #undef ARR_IDX
}



/*!
 * Launch "kernelLocalSolTriPCRDimJ<RealType, unsigned int, unsigned int>"
 * \param [in, out] x                   right-hand side of the system and the solution as the output
 * \param [in]      fact_prev           factorization coefficients of the previous row
 * \param [in]      fact_curr           factorization coefficients of the current row
 * \param [in]      fact_next           factorization coefficients of the next row
 * \param [in]      max_sub_sys_size    maximum sub-system size that can be solved with one memory load
 * \param [in]      Ni                  total number of entries in the i-dimension
 * \param [in]      Nj                  total number of entries in the j-dimension
 * \param [in]      Nk                  total number of entries in the k-dimension
 * \param [in]      arr_stride_i        array access stride in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride in j-dimension, from j to j+1
 * \note array access in k-dimension is contiguous
 */
template<typename RealType>
void localSolTriPCRDimJ(
              RealType*         x,
        const RealType* fact_prev,
        const RealType* fact_curr,
        const RealType* fact_next,
        const unsigned int  max_sub_sys_size,
        const unsigned int Ni, unsigned const int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
) {
    if (Ni <= 256) { // limited by the size of shared memory (16kB)
        constexpr unsigned int BLOCK_SIZE    = 512;
        constexpr unsigned int DATA_SEG      = 64;
        constexpr unsigned int NUM_THREADS_K = DATA_SEG / sizeof(RealType);
        constexpr unsigned int NUM_THREADS_J = BLOCK_SIZE / NUM_THREADS_K;
        const unsigned int smem_size = 2 * DATA_SEG * Nj;
        dim3 grid_size  = {(Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, 1, Ni};
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimJ<RealType), NUM_THREADS_J, NUM_THREADS_K>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimJ<RealType, NUM_THREADS_J, NUM_THREADS_K><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    } else if (Ni <= 512) {
        constexpr unsigned int BLOCK_SIZE    = 1024;
        constexpr unsigned int DATA_SEG      = 32;
        constexpr unsigned int NUM_THREADS_K = DATA_SEG / sizeof(RealType);
        constexpr unsigned int NUM_THREADS_J = BLOCK_SIZE / NUM_THREADS_K;
        const unsigned int smem_size = 2 * DATA_SEG * Nj;
        dim3 grid_size  = {(Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, 1, Ni};
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimJ<RealType), NUM_THREADS_J, NUM_THREADS_K>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimJ<RealType, NUM_THREADS_J, NUM_THREADS_K><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    } else {
        constexpr unsigned int BLOCK_SIZE    = 1024;
        constexpr unsigned int DATA_SEG      = 8;
        constexpr unsigned int NUM_THREADS_K = DATA_SEG / sizeof(RealType);
        constexpr unsigned int NUM_THREADS_J = BLOCK_SIZE / NUM_THREADS_K;
        const unsigned int smem_size = 2 * DATA_SEG * Nj;
        dim3 grid_size  = {(Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, 1, Ni};
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimJ<RealType), NUM_THREADS_J, NUM_THREADS_K>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimJ<RealType, NUM_THREADS_J, NUM_THREADS_K><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    }
}



/*!
 * Solve the local system using PCR along k-dimension
 * \param [in, out] x                   right-hand side of the system and the solution as the output
 * \param [in]      fact_prev           factorization coefficients of the previous row
 * \param [in]      fact_curr           factorization coefficients of the current row
 * \param [in]      fact_next           factorization coefficients of the next row
 * \param [in]      max_sub_sys_size    maximum sub-system size that can be solved with one memory load
 * \param [in]      Ni                  total number of entries in the i-dimension
 * \param [in]      Nj                  total number of entries in the j-dimension
 * \param [in]      Nk                  total number of entries in the k-dimension
 * \param [in]      arr_stride_i        array access stride in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride in j-dimension, from j to j+1
 * \note array access in k-dimension is contiguous
 */
template<typename RealType, unsigned int NUM_THREADS_K, unsigned int NUM_THREADS_J> __global__
void kernelLocalSolTriPCRDimK(
              RealType* __restrict__         x,
        const RealType* __restrict__ fact_prev,
        const RealType* __restrict__ fact_curr,
        const RealType* __restrict__ fact_next,
        const unsigned int      max_sub_sys_size,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
) {
    #define IDX_I             blockIdx.z
    #define IDX_J            ((blockIdx.y * NUM_THREADS_J) + (threadIdx.x / NUM_THREADS_K))
    #define TID_J            (threadIdx.x / NUM_THREADS_K) 
    #define TID_K            (threadIdx.x % NUM_THREADS_K) 
    #define NUM_SYS           Nk
    #define SMEM_IDX(K, J)   ((K) + (J) * Nk)
    #define ARR_IDX(I, J, K) ((I) * arr_stride_i + (J) * arr_stride_j + (K))

    assert(blockIdx.x == 0);
    assert((NUM_THREADS_K * NUM_THREADS_J) == blockDim.x);

    extern __shared__ char smem_general[];
    RealType* smem_realtype = reinterpret_cast<RealType*>(smem_general);
    RealType* smem_buf_0 = &smem_realtype[0];
    RealType* smem_buf_1 = &smem_realtype[NUM_SYS * NUM_THREADS_J];
    RealType* x_read  = smem_buf_0;
    RealType* x_write = smem_buf_1;
    const bool VALID_IJ = (IDX_I < Ni) && (IDX_J < Nj);

    // Load data to shared memory from global memory
    unsigned int idx_k = TID_K;
    while (idx_k < Nk) {
        if (VALID_IJ) x_read[SMEM_IDX(idx_k, TID_J)] = x[ARR_IDX(IDX_I, IDX_J, idx_k)];
        idx_k += NUM_THREADS_K;
    }
    __syncthreads();

    // Parallel cyclic-reduction on shared memory
    int stride = 1;
    int level  = 0;
    while (stride < NUM_SYS) {
        x_read  = (level & 0b1) ? smem_buf_1 : smem_buf_0;
        x_write = (level & 0b1) ? smem_buf_0 : smem_buf_1;

        idx_k = TID_K;
        while (idx_k < Nk) {
            if (VALID_IJ) {
                const int k_sub = idx_k >> level;
                const int n_sub = (NUM_SYS + stride - 1 - (idx_k & (stride-1))) >> level;
                const int row_label = 0b010
                                    + ((k_sub >        0 ) << 2)
                                    + ((k_sub < (n_sub-1))     );
                const unsigned int idx_fact = level * NUM_SYS + locFactIdx<unsigned int>(idx_k, NUM_SYS, stride, max_sub_sys_size);
                const unsigned int SMEM_IDX_CURR = SMEM_IDX(idx_k, TID_J);
                x_write[SMEM_IDX_CURR] = fact_curr[idx_fact] * x_read[SMEM_IDX_CURR];
                if (row_label & 0b100) x_write[SMEM_IDX_CURR] += fact_prev[idx_fact] * x_read[SMEM_IDX(idx_k -  stride, TID_J)];
                if (row_label & 0b001) x_write[SMEM_IDX_CURR] += fact_next[idx_fact] * x_read[SMEM_IDX(idx_k +  stride, TID_J)];
            }
            idx_k += NUM_THREADS_K;
        }
        __syncthreads();
        stride <<= 1;
        level ++;
    }
     
    // Write result to global memeory
    idx_k = TID_K;
    while (idx_k < Nk) {
        if (VALID_IJ) x[ARR_IDX(IDX_I, IDX_J, idx_k)] = x_write[SMEM_IDX(idx_k, TID_J)];
        idx_k += NUM_THREADS_K;
    }
     
    #undef IDX_I
    #undef IDX_J
    #undef TID_J
    #undef TID_K
    #undef NUM_SYS
    #undef SMEM_IDX
    #undef ARR_IDX
}



/*!
 * Launch "kernelLocalSolTriPCRDimK<RealType, unsigned int, unsigned int>"
 * \param [in, out] x                   right-hand side of the system and the solution as the output
 * \param [in]      fact_prev           factorization coefficients of the first previous row
 * \param [in]      fact_curr           factorization coefficients of the current row
 * \param [in]      fact_next           factorization coefficients of the first next row
 * \param [in]      max_sub_sys_size    maximum sub-system size that can be solved with one memory load
 * \param [in]      Ni                  total number of entries in the i-dimension
 * \param [in]      Nj                  total number of entries in the j-dimension
 * \param [in]      Nk                  total number of entries in the k-dimension
 * \param [in]      arr_stride_i        array access stride in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride in j-dimension, from j to j+1
 * \note array access in k-dimension is contiguous
 */
template<typename RealType>
void localSolTriPCRDimK(
              RealType*         x,
        const RealType* fact_prev,
        const RealType* fact_curr,
        const RealType* fact_next,
        const unsigned int  max_sub_sys_size,
        const unsigned int Ni, unsigned const int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
) {
    if (Nk <= 256) {
        constexpr unsigned int NUM_THREADS_K = 128;
        constexpr unsigned int NUM_THREADS_J =  2 * sizeof(double) / sizeof(RealType); 
        constexpr unsigned int BLOCK_SIZE    = NUM_THREADS_K * NUM_THREADS_J;
        const unsigned int smem_size = 2 * Nk * NUM_THREADS_J * sizeof(RealType);
        dim3 grid_size  = dim3(1, (Nj + NUM_THREADS_J - 1) / NUM_THREADS_J, Ni);
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimK<RealType), NUM_THREADS_K, NUM_THREADS_J>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimK<RealType, NUM_THREADS_K, NUM_THREADS_J><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    } else if (Nk <= 512) {
        constexpr unsigned int NUM_THREADS_K = 256;
        constexpr unsigned int NUM_THREADS_J = 2 * sizeof(double) / sizeof(RealType); 
        constexpr unsigned int BLOCK_SIZE    = NUM_THREADS_K * NUM_THREADS_J;
        const unsigned int smem_size = 2 * Nk * NUM_THREADS_J * sizeof(RealType);
        dim3 grid_size  = dim3(1, (Nj + NUM_THREADS_J - 1) / NUM_THREADS_J, Ni);
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimK<RealType), NUM_THREADS_K, NUM_THREADS_J>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimK<RealType, NUM_THREADS_K, NUM_THREADS_J><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    } else {
        constexpr unsigned int NUM_THREADS_K = 1024;
        constexpr unsigned int NUM_THREADS_J = sizeof(double) / sizeof(RealType); 
        constexpr unsigned int BLOCK_SIZE    = NUM_THREADS_K * NUM_THREADS_J;
        const unsigned int smem_size = 2 * Nk * NUM_THREADS_J * sizeof(RealType);
        dim3 grid_size  = dim3(1, (Nj + NUM_THREADS_J - 1) / NUM_THREADS_J, Ni);
        //hipFuncSetAttribute(reinterpret_cast<const void*>(kernelLocalSolTriPCRDimK<RealType), NUM_THREADS_K, NUM_THREADS_J>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
        kernelLocalSolTriPCRDimK<RealType, NUM_THREADS_K, NUM_THREADS_J><<<grid_size, BLOCK_SIZE, smem_size>>>
            (x, fact_prev, fact_curr, fact_next, max_sub_sys_size, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
    }
}



/*!
 * Reduce the current distributed block using blocks from both sides
 * \tparam RealType                 real-value type of flattened local block 
 * \tparam RealTypeComm             real-value type of communicated neighboring block
 * \param [in, out] x_curr          local block as input and eliminated local block as output
 * \param [in]      x_prev_buf      communicated neighboring block from the previous (strided) rank
 * \param [in]      x_next_buf      communicated neighboring block from the next (strided) rank
 * \param [in]      fact_curr       distributed factorization coefficients of the current block
 * \param [in]      fact_prev       distributed factorization coefficients of the previous block
 * \param [in]      fact_next       distributed factorization coefficients of the next block
 * \param [in]      N_batch         number of entries in the non-solve dimension
 * \note solution is stored in column-major with size 2 x N_batch
 */
template<typename RealType, typename RealTypeComm = RealType> __global__
void kernelReduceCurrBlockSymm(
              RealType*     __restrict__ x_curr,
        const RealTypeComm* __restrict__ x_prev_buf,
        const RealTypeComm* __restrict__ x_next_buf,
        const RealType                   fact_curr,
        const RealType                   fact_prev,
        const RealType                   fact_next,
        const unsigned int N_batch)
{
    #define IDX_J        (blockIdx.x * blockDim.x + threadIdx.x)
    #define TID_J                                   threadIdx.x
    #define NUM_THREADS   blockDim.x

    const bool VALID_ENTRY = IDX_J < N_batch;
    extern __shared__ char sm_general[];
    RealType*     sm_curr = reinterpret_cast<RealType*    >(&sm_general[0]);
    RealTypeComm* sm_prev = reinterpret_cast<RealTypeComm*>(&sm_general[NUM_THREADS *  sizeof(RealType)]);
    RealTypeComm* sm_next = reinterpret_cast<RealTypeComm*>(&sm_general[NUM_THREADS * (sizeof(RealType) + sizeof(RealTypeComm))]);

    if (VALID_ENTRY) {
        sm_curr[TID_J] = x_curr    [IDX_J];
        sm_prev[TID_J] = x_prev_buf[IDX_J];
        sm_next[TID_J] = x_next_buf[IDX_J];
    }

    __syncthreads();
    if (VALID_ENTRY) {
        x_curr[IDX_J] = fact_curr * sm_curr[TID_J]
                      + fact_prev * sm_prev[TID_J]
                      + fact_next * sm_next[TID_J];
    }

    #undef IDX_J
    #undef TID_J
    #undef NUM_THREADS
}



/*!
 * Launch kernel "kernelReduceCurrBlockSymm"
 * \tparam RealType                 real-value type of flattened local block 
 * \tparam RealTypeComm             real-value type of communicated neighboring block
 * \param [in, out] x_curr          local block as input and eliminated local block as output
 * \param [in]      x_prev_buf      communicated neighboring block from the previous (strided) rank
 * \param [in]      x_next_buf      communicated neighboring block from the next (strided) rank
 * \param [in]      fact_curr       distributed factorization coefficients of the current block
 * \param [in]      fact_prev       distributed factorization coefficients of the previous block
 * \param [in]      fact_next       distributed factorization coefficients of the next block
 * \param [in]      N_batch         number of entries in the non-solve dimension
 * \note solution is stored in column-major with size 2 x N_batch
 */
template<typename RealType, typename RealTypeComm = RealType>
void reduceCurrBlockSymm(
              RealType*     x_curr,
        const RealTypeComm* x_prev_buf,
        const RealTypeComm* x_next_buf,
        const RealType      fact_curr,
        const RealType      fact_prev,
        const RealType      fact_next,
        const unsigned int N_batch)
{
    unsigned int num_threads_j = 256;
    while (num_threads_j > N_batch) num_threads_j >>= 1;
    int block_size = num_threads_j;
    int  grid_size = (N_batch + num_threads_j - 1) / num_threads_j;
    unsigned int sm_size = num_threads_j * (sizeof(RealType) + 2 * sizeof(RealTypeComm));
    kernelReduceCurrBlockSymm<RealType, RealTypeComm><<<grid_size, block_size, sm_size>>>(x_curr, x_prev_buf, x_next_buf, fact_curr, fact_prev, fact_next, N_batch);
}



/*!
 * Reduce the current distributed block using blocks from one side
 * \tparam RealType                 real-value type of flattened local block 
 * \tparam RealTypeComm             real-value type of communicated neighboring block
 * \param [in, out] x_cur           local block as input and eliminated local block as output
 * \param [in]      x_nbr_buf       communicated neighboring block from the previous (strided) rank
 * \param [in]      fact_cur        distributed factorization coefficients of the current block
 * \param [in]      fact_nbr        distributed factorization coefficients of the previous block
 * \param [in]      N_batch         number of entries in the non-solve dimension
 * \note solution is stored in column-major with size 2 x N_batch
 */
template<typename RealType, typename RealTypeComm = RealType> __global__
void kernelReduceCurrBlockOneSide(
              RealType*     __restrict__ x_cur,
        const RealTypeComm* __restrict__ x_nbr_buf,
        const RealType                   fact_cur,
        const RealType                   fact_nbr,
        const unsigned int N_batch)
{
    // The index of the system is (i,j) and the solution is in column-major
    #define IDX_J        (blockIdx.x * blockDim.x + threadIdx.x)
    #define TID_J                                   threadIdx.x
    #define NUM_THREADS   blockDim.x

    const bool VALID_ENTRY = IDX_J < N_batch;
    extern __shared__ char sm_general[];
    RealType*     sm_cur = reinterpret_cast<RealType*    >(&sm_general[0]);
    RealTypeComm* sm_nbr = reinterpret_cast<RealTypeComm*>(&sm_general[NUM_THREADS * sizeof(RealType)]);

    if (VALID_ENTRY) {
        sm_cur[TID_J] = x_cur    [IDX_J];
        sm_nbr[TID_J] = x_nbr_buf[IDX_J];
    }

    __syncthreads();
    if (VALID_ENTRY) {
        x_cur[IDX_J] = fact_cur * sm_cur[TID_J]
                     + fact_nbr * sm_nbr[TID_J];
    }

    #undef IDX_J
    #undef TID_J
    #undef NUM_THREADS
}



/*!
 * Launch kernel "kernelReduceCurrBlockOneSide"
 * \tparam RealType                 real-value type of flattened local block 
 * \tparam RealTypeComm             real-value type of communicated neighboring block
 * \param [in, out] x_cur           local block as input and eliminated local block as output
 * \param [in]      x_nbr_buf       communicated neighboring block from the previous (strided) rank
 * \param [in]      fact_cur        distributed factorization coefficients of the current block
 * \param [in]      fact_nbr        distributed factorization coefficients of the previous block
 * \param [in]      N_batch         number of entries in the non-solve dimension
 * \note solution is stored in column-major with size 2 x N_batch
 */
template<typename RealType, typename RealTypeComm = RealType>
void reduceCurrBlockOneSide(
              RealType*     x_cur,
        const RealTypeComm* x_nbr_buf,
        const RealType      fact_cur,
        const RealType      fact_nbr,
        const unsigned int N_batch)
{
    unsigned int num_threads_j = 256;
    while (num_threads_j > N_batch) num_threads_j >>= 1;
    dim3 block_size = dim3(num_threads_j);
    dim3  grid_size = dim3((N_batch + num_threads_j - 1) / num_threads_j);
    unsigned int sm_size = num_threads_j * (sizeof(RealType) + sizeof(RealTypeComm));
    kernelReduceCurrBlockOneSide<RealType, RealTypeComm><<<grid_size, block_size, sm_size>>>(x_cur, x_nbr_buf, fact_cur, fact_nbr, N_batch);
}



/*!
 * Locally calcualte the right-hand side of the reduced system, b_hat.
 * \param [in, out]    x_tilde        b_tilde as input and b_hat as output (see Eq (21) in Song et al. JCP (2022) 111443)
 * \param [in]         y_prev_tail    The last two rows of y_{i-1} defined in Eq(17) in Song et al. JCP (2022) 111443
 * \param [in]         y_curr_head    The first two rows of y_{i} defined in Eq(17) in Song et al. JCP (2022) 111443
 * \param [in]         Li_tilde_tail  The non-zero entries in Li_tilde stored in row-major (see Fig.4 in Song et al. JCP (2022) 111443)
 * \param [in]         Ui_tilde_head  The non-zero entries in Ui_tilde stored in row-major (see Fig.4 in Song et al. JCP (2022) 111443)
 * \param [in]         N_batch        Number of entries in the non-solve direction
 * \note The non-solve direction of x_tilde, y_prev_tail and y_curr_head are mapped to contiguous memory layout
 */
template<typename RealType> __global__
void kernelCalcReducedSystemRHSLocal(
              RealType* __restrict__ x_tilde,
        const RealType* __restrict__ y_prev_tail,
        const RealType* __restrict__ y_curr_head,
        const RealType               Li_tilde_tail,
        const RealType               Ui_tilde_head,
        const unsigned int N_batch
    )
{
    const unsigned int      j  = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ char sm_general[];
    RealType* sm_y_prev = reinterpret_cast<RealType*>(&sm_general[0]);
    RealType* sm_y_curr = reinterpret_cast<RealType*>(&sm_general[(1+blockDim.x) * sizeof(RealType)]);

    if (j < N_batch) {
        sm_y_prev[threadIdx.x] = y_prev_tail[j];
        sm_y_curr[threadIdx.x] = y_curr_head[j];
    }

    __syncthreads();

    if (j < N_batch) {
        x_tilde[j] -= Li_tilde_tail * sm_y_prev[threadIdx.x] + Ui_tilde_head * sm_y_curr[threadIdx.x];
    }
}



/*!
 * Launch "kernelCalcReducedSystemRHSLocal"
 * \param [in, out]    x_tilde        b_tilde as input and b_hat as output (see Eq (21) in Song et al. JCP (2022) 111443)
 * \param [in]         y_prev_tail    The last two rows of y_{i-1} defined in Eq(17) in Song et al. JCP (2022) 111443
 * \param [in]         y_curr_head    The first two rows of y_{i} defined in Eq(17) in Song et al. JCP (2022) 111443
 * \param [in]         Li_tilde_tail  The non-zero entries in Li_tilde stored in row-major (see Fig.4 in Song et al. JCP (2022) 111443)
 * \param [in]         Ui_tilde_head  The non-zero entries in Ui_tilde stored in row-major (see Fig.4 in Song et al. JCP (2022) 111443)
 * \param [in]         N_batch        Number of entries in the non-solve direction
 * \note The non-solve direction of x_tilde, y_prev_tail and y_curr_head are mapped to contiguous memory layout
 */
template<typename RealType>
void calcReducedSystemRHSLocal(
              RealType* x_tilde,
        const RealType* y_prev_tail,
        const RealType* y_curr_head,
        const RealType  Li_tilde_tail,
        const RealType  Ui_tilde_head,
        const unsigned int N_batch
    )
{
    unsigned int num_threads_x = 128;
    while(num_threads_x > N_batch) num_threads_x >>= 1;
    unsigned int sm_size = 2 * (num_threads_x + 1) * sizeof(RealType);
    dim3 block_size = dim3(num_threads_x);
    dim3  grid_size = dim3((N_batch + num_threads_x - 1) / num_threads_x);
    kernelCalcReducedSystemRHSLocal<RealType><<<grid_size, block_size, sm_size>>>(x_tilde, y_prev_tail, y_curr_head, Li_tilde_tail, Ui_tilde_head, N_batch);
}



/*!
 * Calculate local solution from y_{rank} in i-dimension using x_tilde from current and next partitions (see Eq.(22) in Song et al. JCP (2022) 111443)
 * \param [in, out]    x_loc            y_{rank} as input and x_{rank} as output (see Fig.4 and Eq.(17) in Song et al. JCP (2022) 111443)
 * \param [in]         x_tilde_curr     The solution to the reduced system in the current partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         x_tilde_next     The solution to the reduced system in the next partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         S                Preprocessed coefficients stored in column-major defined in Eq.(15) in Song et al. JCP (2022) 11143
 * \param [in]         R                Preprocessed coefficients stored in column-major defined in Eq.(16) in Song et al. JCP (2022) 11143
 * \param [in]         Ni               Number of entries in x_loc in i-dimension
 * \param [in]         Nj               Number of entries in x_loc in j-dimension
 * \param [in]         Nk               Number of entries in x_loc in k-dimension
 * \param [in]      arr_stride_i        array access stride of x_loc in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride of x_loc in j-dimension, from j to j+1
 */
template<unsigned int NUM_THREADS_I, unsigned int NUM_THREADS_J, unsigned int NUM_THREADS_K, typename RealType> __global__
void kernelUpdateLocalSolDimI(
              RealType* __restrict__ x_loc,
        const RealType* __restrict__ x_tilde_curr,
        const RealType* __restrict__ x_tilde_next,
        const RealType* __restrict__ S,
        const RealType* __restrict__ R,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
    )
{
    #define TID_I (threadIdx.x / (NUM_THREADS_J * NUM_THREADS_K))
    #define TID_J ((threadIdx.x / NUM_THREADS_K) % NUM_THREADS_J)
    #define TID_K (threadIdx.x % NUM_THREADS_K)
    #define SMEM_IDX(J, K) ((J) * NUM_THREADS_J + (K)) 

    #define IDX_I (blockIdx.z * NUM_THREADS_I + TID_I)
    #define IDX_J (blockIdx.y * NUM_THREADS_J + TID_J)
    #define IDX_K (blockIdx.x * NUM_THREADS_K + TID_K)

    extern __shared__ char smem_general[];
    RealType* smem_x_tilde_curr = reinterpret_cast<RealType*>(&smem_general[0]);
    RealType* smem_x_tilde_next = reinterpret_cast<RealType*>(&smem_general[NUM_THREADS_J * NUM_THREADS_K * sizeof(RealType)]);

    if ((TID_I == 0) && (IDX_J < Nj) && (IDX_K < Nk)) {
        smem_x_tilde_curr[SMEM_IDX(TID_J, TID_K)] = x_tilde_curr[TID_I * Nj * Nk + IDX_J * Nk + IDX_K];
        smem_x_tilde_next[SMEM_IDX(TID_J, TID_K)] = x_tilde_next[TID_I * Nj * Nk + IDX_J * Nk + IDX_K];
    }
    __syncthreads();

    if ((IDX_I < Ni) && (IDX_J < Nj) && (IDX_K < Nk)) {
        x_loc[IDX_I * arr_stride_i + IDX_J * arr_stride_j + IDX_K]
            -= S[IDX_I] * smem_x_tilde_curr[SMEM_IDX(TID_J, TID_K)]
             + R[IDX_I] * smem_x_tilde_next[SMEM_IDX(TID_J, TID_K)];
    }
    
    #undef IDX_I
    #undef IDX_J
    #undef IDX_K
    #undef TID_I
    #undef TID_J
    #undef TID_K
    #undef SMEM_IDX
}



/*!
 * Launch "kernelUpdateLocalSolDimI"
 * \param [in, out]    x_loc            y_{rank} as input and x_{rank} as output (see Fig.4 and Eq.(17) in Song et al. JCP (2022) 111443)
 * \param [in]         x_tilde_curr     The solution to the reduced system in the current partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         x_tilde_next     The solution to the reduced system in the next partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         S                Preprocessed coefficients stored in column-major defined in Eq.(15) in Song et al. JCP (2022) 11143
 * \param [in]         R                Preprocessed coefficients stored in column-major defined in Eq.(16) in Song et al. JCP (2022) 11143
 * \param [in]         Ni               Number of entries in x_loc in i-dimension
 * \param [in]         Nj               Number of entries in x_loc in j-dimension
 * \param [in]         Nk               Number of entries in x_loc in k-dimension
 * \param [in]      arr_stride_i        array access stride of x_loc in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride of x_loc in j-dimension, from j to j+1
 */
template<typename RealType>
void updateLocalSolDimI(
              RealType* x_loc,
        const RealType* x_tilde_curr,
        const RealType* x_tilde_next,
        const RealType* S,
        const RealType* R,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
    )
{
    constexpr unsigned int NUM_THREADS_I = 32;
    constexpr unsigned int NUM_THREADS_J =  1;
    constexpr unsigned int NUM_THREADS_K = 16;
    constexpr unsigned int SMEM_SIZE = 2 * NUM_THREADS_J * NUM_THREADS_K * sizeof(RealType);
    constexpr unsigned int BLOCK_SIZE = NUM_THREADS_I * NUM_THREADS_J * NUM_THREADS_K;
    dim3  grid_size = dim3((Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, (Nj + NUM_THREADS_J - 1) / NUM_THREADS_J, (Ni + NUM_THREADS_I - 1) / NUM_THREADS_I);
    kernelUpdateLocalSolDimI<NUM_THREADS_I, NUM_THREADS_J, NUM_THREADS_K, RealType><<<grid_size, BLOCK_SIZE, SMEM_SIZE>>>
        (x_loc, x_tilde_curr, x_tilde_next, S, R, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
}



/*!
 * Calculate local solution from y_{rank} in j-dimension using x_tilde from current and next partitions (see Eq.(22) in Song et al. JCP (2022) 111443)
 * \param [in, out]    x_loc            y_{rank} as input and x_{rank} as output (see Fig.4 and Eq.(17) in Song et al. JCP (2022) 111443)
 * \param [in]         x_tilde_curr     The solution to the reduced system in the current partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         x_tilde_next     The solution to the reduced system in the next partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         S                Preprocessed coefficients stored in column-major defined in Eq.(15) in Song et al. JCP (2022) 11143
 * \param [in]         R                Preprocessed coefficients stored in column-major defined in Eq.(16) in Song et al. JCP (2022) 11143
 * \param [in]         Ni               Number of entries in x_loc in i-dimension
 * \param [in]         Nj               Number of entries in x_loc in j-dimension
 * \param [in]         Nk               Number of entries in x_loc in k-dimension
 * \param [in]      arr_stride_i        array access stride of x_loc in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride of x_loc in j-dimension, from j to j+1
 */
template<unsigned int NUM_THREADS_I, unsigned int NUM_THREADS_J, unsigned int NUM_THREADS_K, typename RealType> __global__
void kernelUpdateLocalSolDimJ(
              RealType* __restrict__ x_loc,
        const RealType* __restrict__ x_tilde_curr,
        const RealType* __restrict__ x_tilde_next,
        const RealType* __restrict__ S,
        const RealType* __restrict__ R,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
    )
{
    #define SMEM_IDX(I, K) ((I) * NUM_THREADS_K + (K)) 
    extern __shared__ char smem_general[];
    RealType* smem_x_tilde_curr = reinterpret_cast<RealType*>(&smem_general[0]);
    RealType* smem_x_tilde_next = reinterpret_cast<RealType*>(&smem_general[NUM_THREADS_I * NUM_THREADS_K * sizeof(RealType)]);

    #define TID_J (threadIdx.x / (NUM_THREADS_I * NUM_THREADS_K))
    #define TID_I ((threadIdx.x / NUM_THREADS_K) % NUM_THREADS_I)
    #define TID_K (threadIdx.x % NUM_THREADS_K)
    #define IDX_I (blockIdx.z * NUM_THREADS_I + TID_I)
    #define IDX_J (blockIdx.y * NUM_THREADS_J + TID_J)
    #define IDX_K (blockIdx.x * NUM_THREADS_K + TID_K)
    if ((TID_J == 0) && (IDX_I < Ni) && (IDX_K < Nk)) {
        smem_x_tilde_curr[SMEM_IDX(TID_I, TID_K)] = x_tilde_curr[IDX_I * Nk + IDX_K];
        smem_x_tilde_next[SMEM_IDX(TID_I, TID_K)] = x_tilde_next[IDX_I * Nk + IDX_K];
    }
    #undef IDX_I
    #undef IDX_J
    #undef IDX_K
    #undef TID_I
    #undef TID_J
    #undef TID_K
    __syncthreads();

    #define TID_I (threadIdx.x / (NUM_THREADS_J * NUM_THREADS_K))
    #define TID_J ((threadIdx.x / NUM_THREADS_K) % NUM_THREADS_J)
    #define TID_K (threadIdx.x % NUM_THREADS_K)
    #define IDX_I (blockIdx.z * NUM_THREADS_I + TID_I)
    #define IDX_J (blockIdx.y * NUM_THREADS_J + TID_J)
    #define IDX_K (blockIdx.x * NUM_THREADS_K + TID_K)

    if ((IDX_I < Ni) && (IDX_J < Nj) && (IDX_K < Nk)) {
        x_loc[IDX_I * arr_stride_i + IDX_J * arr_stride_j + IDX_K]
            -= S[IDX_J] * smem_x_tilde_curr[SMEM_IDX(TID_I, TID_K)]
             + R[IDX_J] * smem_x_tilde_next[SMEM_IDX(TID_I, TID_K)];
    }
    
    #undef IDX_I
    #undef IDX_J
    #undef IDX_K
    #undef TID_I
    #undef TID_J
    #undef TID_K
    #undef SMEM_IDX
}



/*!
 * Launch "kernelUpdateLocalSolDimJ"
 * \param [in, out]    x_loc            y_{rank} as input and x_{rank} as output (see Fig.4 and Eq.(17) in Song et al. JCP (2022) 111443)
 * \param [in]         x_tilde_curr     The solution to the reduced system in the current partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         x_tilde_next     The solution to the reduced system in the next partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         S                Preprocessed coefficients stored in column-major defined in Eq.(15) in Song et al. JCP (2022) 11143
 * \param [in]         R                Preprocessed coefficients stored in column-major defined in Eq.(16) in Song et al. JCP (2022) 11143
 * \param [in]         Ni               Number of entries in x_loc in i-dimension
 * \param [in]         Nj               Number of entries in x_loc in j-dimension
 * \param [in]         Nk               Number of entries in x_loc in k-dimension
 * \param [in]      arr_stride_i        array access stride of x_loc in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride of x_loc in j-dimension, from j to j+1
 */
template<typename RealType>
void updateLocalSolDimJ(
              RealType* x_loc,
        const RealType* x_tilde_curr,
        const RealType* x_tilde_next,
        const RealType* S,
        const RealType* R,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
    )
{
    constexpr unsigned int NUM_THREADS_I =  1;
    constexpr unsigned int NUM_THREADS_J = 32;
    constexpr unsigned int NUM_THREADS_K = 16;
    constexpr unsigned int SMEM_SIZE     = 2 * NUM_THREADS_I * NUM_THREADS_K * sizeof(RealType);
    constexpr unsigned int BLOCK_SIZE    = NUM_THREADS_I * NUM_THREADS_J * NUM_THREADS_K;
    dim3  grid_size = dim3((Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, (Nj + NUM_THREADS_J - 1) / NUM_THREADS_J, (Ni + NUM_THREADS_I - 1) / NUM_THREADS_I);
    kernelUpdateLocalSolDimJ<NUM_THREADS_I, NUM_THREADS_J, NUM_THREADS_K, RealType><<<grid_size, BLOCK_SIZE, SMEM_SIZE>>>
        (x_loc, x_tilde_curr, x_tilde_next, S, R, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
}



/*!
 * Calculate local solution from y_{rank} in k-dimension using x_tilde from current and next partitions (see Eq.(22) in Song et al. JCP (2022) 111443)
 * \param [in, out]    x_loc            y_{rank} as input and x_{rank} as output (see Fig.4 and Eq.(17) in Song et al. JCP (2022) 111443)
 * \param [in]         x_tilde_curr     The solution to the reduced system in the current partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         x_tilde_next     The solution to the reduced system in the next partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         S                Preprocessed coefficients stored in column-major defined in Eq.(15) in Song et al. JCP (2022) 11143
 * \param [in]         R                Preprocessed coefficients stored in column-major defined in Eq.(16) in Song et al. JCP (2022) 11143
 * \param [in]         Ni               Number of entries in x_loc in i-dimension
 * \param [in]         Nj               Number of entries in x_loc in j-dimension
 * \param [in]         Nk               Number of entries in x_loc in k-dimension
 * \param [in]      arr_stride_i        array access stride of x_loc in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride of x_loc in j-dimension, from j to j+1
 */
template<unsigned int NUM_THREADS_I, unsigned int NUM_THREADS_J, unsigned int NUM_THREADS_K, typename RealType> __global__
void kernelUpdateLocalSolDimK(
              RealType* __restrict__ x_loc,
        const RealType* __restrict__ x_tilde_curr,
        const RealType* __restrict__ x_tilde_next,
        const RealType* __restrict__ S,
        const RealType* __restrict__ R,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
    )
{

    #define SMEM_IDX(I, J) ((I) * NUM_THREADS_J + (J)) 

    extern __shared__ char smem_general[];
    RealType* smem_x_tilde_curr = reinterpret_cast<RealType*>(&smem_general[0]);
    RealType* smem_x_tilde_next = reinterpret_cast<RealType*>(&smem_general[NUM_THREADS_I * NUM_THREADS_J * sizeof(RealType)]);

    #define TID_K (threadIdx.x / (NUM_THREADS_I * NUM_THREADS_J))
    #define TID_I ((threadIdx.x / NUM_THREADS_J) % NUM_THREADS_I)
    #define TID_J (threadIdx.x % NUM_THREADS_J)
    #define IDX_I (blockIdx.z * NUM_THREADS_I + TID_I)
    #define IDX_J (blockIdx.y * NUM_THREADS_J + TID_J)
    #define IDX_K (blockIdx.x * NUM_THREADS_K + TID_K)

    if ((TID_K == 0) && (IDX_I < Ni) && (IDX_J < Nj)) {
        smem_x_tilde_curr[SMEM_IDX(TID_I, TID_J)] = x_tilde_curr[IDX_I * Nj + IDX_J];
        smem_x_tilde_next[SMEM_IDX(TID_I, TID_J)] = x_tilde_next[IDX_I * Nj + IDX_J];
    }
    #undef IDX_I
    #undef IDX_J
    #undef IDX_K
    #undef TID_I
    #undef TID_J
    #undef TID_K
    __syncthreads();

    #define TID_I (threadIdx.x / (NUM_THREADS_J * NUM_THREADS_K))
    #define TID_J ((threadIdx.x / NUM_THREADS_K) % NUM_THREADS_J)
    #define TID_K (threadIdx.x % NUM_THREADS_K)
    #define IDX_I (blockIdx.z * NUM_THREADS_I + TID_I)
    #define IDX_J (blockIdx.y * NUM_THREADS_J + TID_J)
    #define IDX_K (blockIdx.x * NUM_THREADS_K + TID_K)
    if ((IDX_I < Ni) && (IDX_J < Nj) && (IDX_K < Nk)) {
        x_loc[IDX_I * arr_stride_i + IDX_J * arr_stride_j + IDX_K]
            -= S[IDX_K] * smem_x_tilde_curr[SMEM_IDX(TID_I, TID_J)]
             + R[IDX_K] * smem_x_tilde_next[SMEM_IDX(TID_I, TID_J)];
    }
    
    #undef IDX_I
    #undef IDX_J
    #undef IDX_K
    #undef TID_I
    #undef TID_J
    #undef TID_K
    #undef SMEM_IDX
}



/*!
 * Launch "kernelUpdateLocalSolDimK"
 * \param [in, out]    x_loc            y_{rank} as input and x_{rank} as output (see Fig.4 and Eq.(17) in Song et al. JCP (2022) 111443)
 * \param [in]         x_tilde_curr     The solution to the reduced system in the current partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         x_tilde_next     The solution to the reduced system in the next partition (see Fig.4 in Song et al. JCP (2022) 11143)
 * \param [in]         S                Preprocessed coefficients stored in column-major defined in Eq.(15) in Song et al. JCP (2022) 11143
 * \param [in]         R                Preprocessed coefficients stored in column-major defined in Eq.(16) in Song et al. JCP (2022) 11143
 * \param [in]         Ni               Number of entries in x_loc in i-dimension
 * \param [in]         Nj               Number of entries in x_loc in j-dimension
 * \param [in]         Nk               Number of entries in x_loc in k-dimension
 * \param [in]      arr_stride_i        array access stride of x_loc in i-dimension, from i to i+1
 * \param [in]      arr_stride_j        array access stride of x_loc in j-dimension, from j to j+1
 */
template<typename RealType>
void updateLocalSolDimK(
              RealType* x_loc,
        const RealType* x_tilde_curr,
        const RealType* x_tilde_next,
        const RealType* S,
        const RealType* R,
        const unsigned int Ni, const unsigned int Nj, const unsigned int Nk,
        const unsigned int arr_stride_i, const unsigned int arr_stride_j
    )
{
    constexpr unsigned int NUM_THREADS_I =  1;
    constexpr unsigned int NUM_THREADS_J = 16;
    constexpr unsigned int NUM_THREADS_K = 32;
    constexpr unsigned int SMEM_SIZE     = 2 * NUM_THREADS_I * NUM_THREADS_J * sizeof(RealType);
    constexpr unsigned int BLOCK_SIZE    = NUM_THREADS_I * NUM_THREADS_J * NUM_THREADS_K;
    dim3  grid_size = dim3((Nk + NUM_THREADS_K - 1) / NUM_THREADS_K, (Nj + NUM_THREADS_J - 1) / NUM_THREADS_J, (Ni + NUM_THREADS_I - 1) / NUM_THREADS_I);
    kernelUpdateLocalSolDimK<NUM_THREADS_I, NUM_THREADS_J, NUM_THREADS_K, RealType><<<grid_size, BLOCK_SIZE, SMEM_SIZE>>>
        (x_loc, x_tilde_curr, x_tilde_next, S, R, Ni, Nj, Nk, arr_stride_i, arr_stride_j);
}






// EXPLICIT INSTANTIATION
template void localSolTriPCRDimI<double>(double*, const double*, const double*, const double*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void localSolTriPCRDimI< float>( float*, const  float*, const  float*, const  float*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void localSolTriPCRDimJ<double>(double*, const double*, const double*, const double*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void localSolTriPCRDimJ< float>( float*, const  float*, const  float*, const  float*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void localSolTriPCRDimK<double>(double*, const double*, const double*, const double*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void localSolTriPCRDimK< float>( float*, const  float*, const  float*, const  float*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);

template void reduceCurrBlockSymm<double, double>(double*, const double*, const double*, const double, const double, const double, const unsigned int);
template void reduceCurrBlockSymm<double,  float>(double*, const  float*, const  float*, const double, const double, const double, const unsigned int);

template void reduceCurrBlockOneSide<double, double>(double*, const double*, const double, const double, const unsigned int);
template void reduceCurrBlockOneSide<double,  float>(double*, const  float*, const double, const double, const unsigned int);

template void calcReducedSystemRHSLocal<double>(double*, const double*, const double*, const double, const double, const unsigned int);
template void calcReducedSystemRHSLocal< float>( float*, const  float*, const  float*, const  float, const  float, const unsigned int);

template void updateLocalSolDimI<double>(double*, const double*, const double*, const double*, const double*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void updateLocalSolDimI< float>( float*, const  float*, const  float*, const  float*, const  float*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void updateLocalSolDimJ<double>(double*, const double*, const double*, const double*, const double*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void updateLocalSolDimJ< float>( float*, const  float*, const  float*, const  float*, const  float*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void updateLocalSolDimK<double>(double*, const double*, const double*, const double*, const double*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);
template void updateLocalSolDimK< float>( float*, const  float*, const  float*, const  float*, const  float*, const unsigned int, const unsigned int, const unsigned int, const unsigned int, const unsigned int);

} // namespace tri
} // namespace cmpk
